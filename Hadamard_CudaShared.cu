#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

void dump_arr(size_t numCol, size_t numRow, float* Arr)
{
    for (size_t j = 0; j < numCol; ++j) {
      for (size_t i = 0; i < numRow; ++i) {
      printf("%.2f ", Arr[i * numCol + j]);
    }
    printf("\n");
  }
}

__global__
void cuda_hadamard_sharedv1(size_t numCol, size_t numRow, float* Z, float* X, float* Y)
{
  // https://stackoverflow.com/questions/7903566/how-is-2d-shared-memory-arranged-in-cuda
  __shared__ float sh_X[TILE_WIDTH][TILE_WIDTH];
  __shared__ float sh_Y[TILE_WIDTH][TILE_WIDTH];

  // map from threadIdx/BlockIdx to data position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  // calculate the global id into the one dimensional array
  int gid = x + y * numCol;

  // load shared memory
  sh_X[threadIdx.y][threadIdx.x] = X[threadIdx.x * numCol + threadIdx.y];
  sh_Y[threadIdx.y][threadIdx.x] = Y[threadIdx.x * numCol + threadIdx.y];

  // synchronize threads not really needed but keep it for convenience
  __syncthreads();

  // write data back to global memory
  Z[gid] = sh_X[threadIdx.y][threadIdx.x] * sh_Y[threadIdx.y][threadIdx.x];
}



int main()
{
  const size_t ARRAY_DIM = 4096;
  const size_t ARRAY_BYTES = ARRAY_DIM * ARRAY_DIM * sizeof(float);
  size_t NUM_EXEC = 30;

  // Array Initialization
  float* X;
  float* Y;
  float* Z;

  // Array Malloc
  hipMallocManaged(&X, ARRAY_BYTES);
  hipMallocManaged(&Y, ARRAY_BYTES);
  hipMallocManaged(&Z, ARRAY_BYTES);

  // get gpu ID
  int device = -1;
  hipGetDevice(&device);

  // Mem advise
  hipMemAdvise(X, ARRAY_BYTES, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(X, ARRAY_BYTES, hipMemAdviseSetReadMostly, hipCpuDeviceId);
  hipMemAdvise(Y, ARRAY_BYTES, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(Y, ARRAY_BYTES, hipMemAdviseSetReadMostly, hipCpuDeviceId);

  // "prefetch data" to create CPU page memory
  hipMemPrefetchAsync(X, ARRAY_BYTES, hipCpuDeviceId, NULL);
  // "prefetch data" to create CPU page memory
  hipMemPrefetchAsync(Y, ARRAY_BYTES, hipCpuDeviceId, NULL);
  // "prefetch data" to create GPU page memory
  hipMemPrefetchAsync(Z, ARRAY_BYTES, device, NULL);

  // initialize array contents
  for (size_t i = 0; i < ARRAY_DIM; ++i) {
    for (size_t j = 0; j < ARRAY_DIM; ++j) {
      X[ARRAY_DIM * i + j] = 1;
      Y[ARRAY_DIM * i + j] = 5;
    }
  }

  // "Prefetch data" from CPU-GPU
  hipMemPrefetchAsync(X, ARRAY_BYTES, device, NULL);
  hipMemPrefetchAsync(Y, ARRAY_BYTES, device, NULL);

  // setup CUDA kernel
  // https://www.cs.emory.edu/~cheung/Courses/355/Syllabus/94-CUDA/2D-grids.html
  size_t threadDimBlockx = TILE_WIDTH;
  size_t threadDimBlocky = TILE_WIDTH;

  dim3 blockShape = dim3(threadDimBlockx, threadDimBlocky);
  // https://selkie.macalester.edu/csinparallel/modules/GPUProgramming/build/html/CUDA2D/CUDA2D.html
  // https://medium.com/@harsh20111997/cuda-programming-2d-convolution-8476300f566e
  dim3 gridShape = dim3( (ARRAY_DIM + threadDimBlockx - 1) / threadDimBlockx, (ARRAY_DIM + threadDimBlocky - 1)/threadDimBlocky );

  for (size_t i = 0; i < NUM_EXEC; ++i) {
    cuda_hadamard_sharedv1 <<< gridShape, blockShape >>> (ARRAY_DIM, ARRAY_DIM, Z, X, Y);
  }

  hipDeviceSynchronize();

  // "Prefetch data" from GPU-CPU
  hipMemPrefetchAsync(X, ARRAY_BYTES, hipCpuDeviceId, NULL);
  hipMemPrefetchAsync(Y, ARRAY_BYTES, hipCpuDeviceId, NULL);
  hipMemPrefetchAsync(Z, ARRAY_BYTES, hipCpuDeviceId, NULL);


  // error checking
  size_t errCount = 0;

  for (size_t i = 0; i < ARRAY_DIM; ++i) {
    for (size_t j = 0; j <ARRAY_DIM; ++j ) {
      if (X[i * ARRAY_DIM + j] * Y[i * ARRAY_DIM + j] != Z[i * ARRAY_DIM + j]) {
        errCount++;
      }
    }
  }

  printf("Array Dimension: %lux%lu\n", ARRAY_DIM, ARRAY_DIM);
  printf("Thread Block Dimension: %dx%d\n", TILE_WIDTH, TILE_WIDTH);
  printf("Total error count: %lu\n", errCount);

  // dump_arr(ARRAY_DIM, ARRAY_DIM, X);
  printf("\n\n");
  // dump_arr(ARRAY_DIM, ARRAY_DIM, Z);

  // free memory
  hipFree(X);
  hipFree(Y);
  hipFree(Z);


  return 0;
}

